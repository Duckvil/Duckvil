#include "hip/hip_runtime.h"
#include "glm/glm.hpp"

#include "glm/gtx/quaternion.hpp"
#include "glm/gtx/transform.hpp"
#include "glm/gtx/euler_angles.hpp"

#include "Objects.h"

#define EPSILON 1e-21f

#define FWDTFORM true
#define BCKTFORM false

#define MAX_DEPTH 3

using namespace Duckvil::Graphics::RayTracer;

__device__ float clamp(float x, float a, float b)
{
    return max(a, min(b, x));
}

__device__ float3 normalize(float3 vec)
{
    float n = sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);

    return make_float3(vec.x / n, vec.y / n, vec.z / n);
}

__device__ float norm(float3 vec)
{
    return sqrtf(vec.x * vec.x + vec.y * vec.y + vec.z * vec.z);
}

__device__ float3 cross(float3 A, float3 B)
{
    float3 _res = make_float3(0, 0, 0);

    _res.x = A.y * B.z - A.z * B.y;
    _res.y = A.z * B.x - A.x * B.z;
    _res.z = A.x * B.y - A.y * B.x;

    return _res;
}

__device__ float dot(float3 A, float3 B)
{
    float _res = 0;

    _res += A.x * B.x;
    _res += A.y * B.y;
    _res += A.z * B.z;

    return _res;
}

__device__ float3 operator-(float3 vec1, float3 vec2)
{
    return make_float3(vec1.x - vec2.x, vec1.y - vec2.y, vec1.z - vec2.z);
}

__device__ float3 operator+(float3 vec1, float3 vec2)
{
    return make_float3(vec1.x + vec2.x, vec1.y + vec2.y, vec1.z + vec2.z);
}

__device__ float3& operator+=(float3& vec1, const float3& vec2)
{
    vec1.x += vec2.x;
    vec1.y += vec2.y;
    vec1.z += vec2.z;

    return vec1;
}

__device__ float3 operator*(float x, float3 vec)
{
    return make_float3(vec.x * x, vec.y * x, vec.z * x);
}

__device__ float3 operator*(float3 vec, float x)
{
    return make_float3(vec.x * x, vec.y * x, vec.z * x);
}

__device__ unsigned int rgbToInt(float _fR, float _fG, float _fB)
{
	return (int(255)<<24) | (int(_fB * 255.f)<<16) | (int(_fG * 255.f)<<8) | int(_fR * 255.f);
}

// __device__ glm::vec3 operator* (const glm::mat3& lhs, const glm::vec3& rhs)
// {
// 	glm::vec3 result;
	
// 	// Loop over rows and columns and perform the multiplication operation element-by-element.
// 	for (int row=0; row<3; ++row)
// 	{
// 		float cumulativeSum = 0.f;
// 		for (int col=0; col<3; ++col)
// 		{
// 			// cumulativeSum += (lhs.GetElement(row,col) * rhs.GetElement(col));
//             cumulativeSum += lhs[row][col] * rhs[col];
// 		}
// 		// result.SetElement(row, cumulativeSum);
//         result[row] = cumulativeSum;
// 	}
	
// 	return result;
// }

// __device__ float3 intToRgb(unsigned int _color)
// {
//     uint8_t red = (_color & 0x000000FF);
//     uint8_t green = (_color & 0x0000FF00) >> 8;
//     uint8_t blue = (_color & 0x000000FF) >> 16;

// 	return make_float3(red / 255.f, green / 255.f, blue / 255.f);
// }

struct Ray
{
    float3 m_f3Point1;
    float3 m_f3Point2;
    float3 m_f3Lab;

    __device__ Ray()
    {
        m_f3Point1 = make_float3(0, 0, 0);
        m_f3Point2 = make_float3(0, 0, 1);
        m_f3Lab = m_f3Point2 - m_f3Point1;
    }

    __device__ Ray(float3 _f3Point1, float3 _f3Point2) :
        m_f3Point1(_f3Point1),
        m_f3Point2(_f3Point2)
    {
        m_f3Lab = m_f3Point2 - m_f3Point1;
    }
};

struct Transform
{
    glm::mat4 m_mat4ForwardTransform;
    glm::mat4 m_mat4BackwardTransform;
    glm::mat3 m_mat3LinearTransform;

    // __device__ Transform() :
    //     m_mat4ForwardTransform(glm::mat4(1.f)),
    //     m_mat4BackwardTransform(glm::inverse(m_mat4ForwardTransform)),
    //     m_mat3LinearTransform(glm::transpose(m_mat4BackwardTransform))
    // {

    // }

    // __device__ Transform(const glm::mat4& _mat4Forward) :
    //     m_mat4ForwardTransform(_mat4Forward),
    //     m_mat4BackwardTransform(glm::inverse(m_mat4ForwardTransform)),
    //     m_mat3LinearTransform(glm::transpose(m_mat4BackwardTransform))
    // {

    // }

    // __device__ Transform(const glm::mat4& _mat4Forward, const glm::mat4& _mat4Backward) :
    //     m_mat4ForwardTransform(_mat4Forward),
    //     m_mat4BackwardTransform(_mat4Backward),
    //     m_mat3LinearTransform(glm::transpose(m_mat4BackwardTransform))
    // {

    // }
};

// __device__ void transform_extract_linear(Transform& _t)
// {
//     _t.m_mat4LinearTransform = glm::transpose(_t.m_mat4BackwardTransform);
// }

__device__ void transform_set(Transform& _t, const glm::mat4& _mat4Forward, const glm::mat4& _mat4Backward)
{
    _t.m_mat4ForwardTransform = _mat4Forward;
    _t.m_mat4BackwardTransform = _mat4Backward;
    _t.m_mat3LinearTransform = glm::transpose(_mat4Backward);
}

__device__ void transform_set(Transform& _t, const glm::mat4& _mat4Forward)
{
    _t.m_mat4ForwardTransform = _mat4Forward;
    _t.m_mat4BackwardTransform = glm::inverse(_t.m_mat4ForwardTransform);
    _t.m_mat3LinearTransform = glm::transpose(_t.m_mat4BackwardTransform);
}

// __device__ Transform operator*(const Transform& _lhs, const Transform& _rhs)
// {
//     return Transform(_lhs.m_mat4ForwardTransform * _rhs.m_mat4ForwardTransform);
// }

__device__ float3 transform_apply(Transform& _transform, const float3& _input, bool _dirFlag)
{
    glm::vec4 _res;
    glm::vec4 _tmp(
        _input.x,
        _input.y,
        _input.z,
        1.f
    );

    if(_dirFlag)
    {
        _res = _transform.m_mat4ForwardTransform * _tmp;
    }
    else
    {
        _res = _transform.m_mat4BackwardTransform * _tmp;
    }

    return make_float3(_res.x, _res.y, _res.z);
}

__device__ float3 transform_apply_norm(const Transform& _t, const float3& _i)
{
    glm::vec3 _r = _t.m_mat3LinearTransform * glm::vec3(_i.x, _i.y, _i.z);

    return make_float3(_r.x, _r.y, _r.z);
}

__device__ Ray transform_apply(Transform& _transform, const Ray& _inputRay, bool _dirFlag)
{
    Ray _outputRay;

    if(_dirFlag)
    {
        _outputRay.m_f3Point1 = transform_apply(_transform, _inputRay.m_f3Point1, FWDTFORM);
        _outputRay.m_f3Point2 = transform_apply(_transform, _inputRay.m_f3Point2, FWDTFORM);
        _outputRay.m_f3Lab = _outputRay.m_f3Point2 - _outputRay.m_f3Point1;
    }
    else
    {
        _outputRay.m_f3Point1 = transform_apply(_transform, _inputRay.m_f3Point1, BCKTFORM);
        _outputRay.m_f3Point2 = transform_apply(_transform, _inputRay.m_f3Point2, BCKTFORM);
        _outputRay.m_f3Lab = _outputRay.m_f3Point2 - _outputRay.m_f3Point1;
    }

    return _outputRay;
}

// struct Camera
// {
// 	float3 m_f3Position;
//     float3 m_f3LookAt;
//     float3 m_f3Up;
//     float m_fLength;
//     float m_fHorizontalSize;
//     float m_fAspectRatio;

//     float3 m_f3AlignmentVector;
//     float3 m_f3ProjectionScreenU;
//     float3 m_f3ProjectionScreenV;
//     float3 m_f3ProjectionScreenCentre;
// };

struct Hit
{
    float3 m_f3POI;
    float3 m_f3POI_N;
    float3 m_f3POI_C;
};

__device__ bool generate_ray(Camera& _camera, Ray& _ray, float x, float y)
{
    float3 _screenWorldPart1 = _camera.m_f3ProjectionScreenCentre + (_camera.m_f3ProjectionScreenU * x);
    float3 _screenWorldCoord = _screenWorldPart1 + (_camera.m_f3ProjectionScreenV * y);

    _ray.m_f3Point1 = _camera.m_f3Position;
    _ray.m_f3Point2 = _screenWorldCoord;
    _ray.m_f3Lab = _screenWorldCoord - _camera.m_f3Position;

    return true;
}

__device__ bool test(Sphere& _object, const Ray& _ray, float3& _f3Intersection, float3& _f3Normal, float3& _f3Color)
{
    float3 _vhat = normalize(_ray.m_f3Lab);
    float b = 2.f * dot(_ray.m_f3Point1, _vhat);
    float c = dot(_ray.m_f3Point1, _ray.m_f3Point1) - 1.f;
    float intTest = (b * b) - 4.f * c;

    if(intTest <= 0.f)
    {
        return false;
    }

    float numSqrt = sqrtf(intTest);
    float t1 = (-b + numSqrt) / 2.f;
    float t2 = (-b - numSqrt) / 2.f;

    if((t1 < 0) || (t2 < 0))
    {
        return false;
    }

    if(t1 < t2)
    {
        if(t1 > 0.f)
        {
            _f3Intersection = _ray.m_f3Point1 + (_vhat * t1);
        }
        else
        {
            if(t2 > 0.f)
            {
                _f3Intersection = _ray.m_f3Point1 + (_vhat * t2);
            }
            else
            {
                return false;
            }
        }
    }
    else
    {
        if(t2 > 0.f)
        {
            _f3Intersection = _ray.m_f3Point1 + (_vhat * t2);
        }
        else
        {
            if(t1 > 0.f)
            {
                _f3Intersection = _ray.m_f3Point1 + (_vhat * t1);
            }
            else
            {
                return false;
            }
        }
    }

    _f3Color = _object.m_f3Color;

    return true;
}

__device__ bool test(Plane& _object, const Ray& _ray, float3& _f3Intersection, float3& _f3Normal, float3& _f3Color)
{
    float3 _k = normalize(_ray.m_f3Lab);

    if(fabs(_k.z - 0.f) < EPSILON)
    {
        return false;
    }

    float _t = _ray.m_f3Point1.z / -_k.z;

    if(_t <= 0.f)
    {
        return false;
    }

    float _u = _ray.m_f3Point1.x + (_k.x * _t);
    float _v = _ray.m_f3Point1.y + (_k.y * _t);

    if((abs(_u) >= 1.f) || (abs(_v) >= 1.f))
    {
        return false;
    }

    _f3Intersection = _ray.m_f3Point1 + _t * _k;
    _f3Color = _object.m_f3Color;

    return true;
}

__device__ bool test(ObjectStream* _pObjectStream, unsigned int _uiStreamOffset, ObjectChunk** _ppCurrentObject, const Ray& _ray, float3& _f3Intersection, float3& _f3Normal, float3& _f3Color, ObjectChunk* _pCurrentObject = nullptr)
{
    unsigned char* _curr = reinterpret_cast<unsigned char*>(_pObjectStream);

    ObjectStream* _stream = reinterpret_cast<ObjectStream*>(_curr);
    _curr += sizeof(ObjectStream) + _uiStreamOffset;

    ObjectChunk* _chunk = reinterpret_cast<ObjectChunk*>(_curr);
    _curr += sizeof(ObjectChunk);

    *_ppCurrentObject = _chunk;

    if(_chunk == _pCurrentObject)
    {
        return false;
    }

    if(_chunk->m_material != MaterialType::none)
    {
        _curr += sizeof(SpecularMaterial);
    }

    Transform* _transform = reinterpret_cast<Transform*>(_curr);
    _curr += sizeof(Transform);

    Ray _bckRay = transform_apply(*_transform, _ray, BCKTFORM);

    if(_chunk->m_type == ObjectType::sphere)
    {
        Sphere* _object = reinterpret_cast<Sphere*>(_curr);
        _curr += sizeof(Sphere);

        float3 _poi;

        if(!test(*_object, _bckRay, _poi, _f3Normal, _f3Color))
        {
            return false;
        }

        _f3Intersection = transform_apply(*_transform, _poi, FWDTFORM);
        // _f3Normal = normalize(transform_apply_norm(*_transform, _poi));

        float3 _newOrigin = transform_apply(*_transform, make_float3(0, 0, 0), FWDTFORM);

        _f3Normal = normalize(_f3Intersection - _newOrigin);

        return true;
    }
    else if(_chunk->m_type == ObjectType::plane)
    {
        Plane* _object = reinterpret_cast<Plane*>(_curr);
        _curr += sizeof(Plane);

        float3 _poi;

        if(!test(*_object, _bckRay, _poi, _f3Normal, _f3Color))
        {
            return false;
        }

        _f3Intersection = transform_apply(*_transform, _poi, FWDTFORM);

        float3 _localOrigin = make_float3(0, 0, 0);
        float3 _normal = make_float3(0, 0, -1);
        float3 _globalOrigin = transform_apply(*_transform, _localOrigin, FWDTFORM);

        _f3Normal = normalize(transform_apply(*_transform, _normal, FWDTFORM) - _globalOrigin);

        // float3 _normal = make_float3(0, 0, -1);

        // _f3Normal = normalize(transform_apply_norm(*_transform, _normal));

        return true;
    }

    return false;
}

__device__ bool find_closest(ObjectStream* _pObjectStream, const Ray& _ray, ObjectChunk** _ppClosestObject, float3& _f3POI, float3& _f3POI_N, float3& _f3POI_C)
{
    unsigned int _currentObjectStreamOffset = 0;
    uint32_t _objectStreamSize = _pObjectStream->m_uiSize;

    float _minDist = 1e6;
    bool _intFound = false;

    float3 _intPoint = make_float3(0, 0, 0);
    float3 _localNormal = make_float3(0, 0, 0);
    float3 _localColor = make_float3(0, 0, 0);

    ObjectChunk* _currentObject;

    while(_currentObjectStreamOffset < _objectStreamSize)
    {
        if(test(_pObjectStream, _currentObjectStreamOffset, &_currentObject, _ray, _intPoint, _localNormal, _localColor))
        {
            _intFound = true;

            float _dist = norm(_intPoint - _ray.m_f3Point1);

            if(_dist >= _minDist)
            {
                continue;
            }

            _minDist = _dist;

            *_ppClosestObject = _currentObject;

            _f3POI = _intPoint;
            _f3POI_N = _localNormal;
            _f3POI_C = _localColor;
        }

        _currentObjectStreamOffset += sizeof(ObjectChunk) + _currentObject->m_uiSize;
    }

    return _intFound;
}

__device__ bool compute_light(PointLight& _light, const float3& _f3IntPoint, const float3& _f3LocalNormal, ObjectStream* _pObjectStream, ObjectChunk* _pCurrentObject, float3& _f3Color, float& _fIntensity)
{
    float3 _dir = normalize(_light.m_f3Location - _f3IntPoint);
    float _lightDist = norm(_light.m_f3Location - _f3IntPoint);
    float3 _startPoint = _f3IntPoint;

    float3 _poi;
    float3 _poiNormal;
    float3 _poiColor;

    unsigned int _currentObjectStreamOffset = 0;
    uint32_t _objectStreamSize = _pObjectStream->m_uiSize;

    Ray _lightRay(_startPoint, _startPoint + _dir);

    bool _validInt = false;

    ObjectChunk* _currentObject;

    while(_currentObjectStreamOffset < _objectStreamSize)
    {
        if(
            test(_pObjectStream, _currentObjectStreamOffset, &_currentObject, _lightRay, _poi, _poiNormal, _f3Color, _pCurrentObject) &&
            // _currentObject != _pCurrentObject &&
            !(norm(_poi - _startPoint) > _lightDist)
        )
        {
            _validInt = true;

            break;
        }

        _currentObjectStreamOffset += sizeof(ObjectChunk) + _currentObject->m_uiSize;
    }

    _f3Color = _light.m_f3Color;

    if(_validInt)
    {
        _fIntensity = 0.f;

        return false;
    }

    float _angle = acos(dot(_f3LocalNormal, _dir));

    if(_angle > 1.5708)
    {
        _fIntensity = 0.f;

        return false;
    }

    _fIntensity = _light.m_fIntensity * (1.f - (_angle / 1.5708));

    return true;
}

__device__ bool compute_light(LightStream* _pLightStream, unsigned int _uiLightChunkOffset, const float3& _f3IntPoint, const float3& _f3LocalNormal, ObjectStream* _pObjectStream, ObjectChunk* _pCurrentObject, float3& _f3Color, float& _fIntensity)
{
    unsigned char* _curr = reinterpret_cast<unsigned char*>(_pLightStream);

    LightStream* _lightStream = reinterpret_cast<LightStream*>(_curr);
    _curr += sizeof(LightStream) + _uiLightChunkOffset;

    LightChunk* _chunk = reinterpret_cast<LightChunk*>(_curr);
    _curr += sizeof(LightChunk);

    if(_chunk->m_type == LightType::point_light)
    {
        PointLight* _light = reinterpret_cast<PointLight*>(_curr);
        _curr += sizeof(PointLight);

        return compute_light(*_light, _f3IntPoint, _f3LocalNormal, _pObjectStream, _pCurrentObject, _f3Color, _fIntensity);
    }

    return false;
}

__device__ float3 g_f3AmbientColor = { 1.f, 1.f, 1.f };
__device__ float g_fAmbientIntensity = 0.2f;

__device__ float3 material_compute_diffuse_color(ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const float3& _f3BaseColor)
{
    float3 _diffuseColor = make_float3(0, 0, 0);

    float _intensity;
    float3 _color;
    bool _illumFound = false;

    unsigned int _currentLightStreamSize = 0;
    uint32_t _lightStreamSize = _pLightStream->m_uiSize;

    while(_currentLightStreamSize < _lightStreamSize)
    {
        if(compute_light(_pLightStream, _currentLightStreamSize, _f3IntPoint, _f3LocalNormal, _pObjectStream, _pCurrentObject, _color, _intensity))
        {
            _illumFound = true;

            _diffuseColor.x += _color.x * _intensity;
            _diffuseColor.y += _color.y * _intensity;
            _diffuseColor.z += _color.z * _intensity;
        }

        _currentLightStreamSize += sizeof(LightChunk) + sizeof(PointLight);
    }

    if(_illumFound)
    {
        _diffuseColor.x *= _f3BaseColor.x;
        _diffuseColor.y *= _f3BaseColor.y;
        _diffuseColor.z *= _f3BaseColor.z;
    }
    // else
    // {
    //     _diffuseColor.x = (g_f3AmbientColor.x * g_fAmbientIntensity) * _f3BaseColor.x;
    //     _diffuseColor.y = (g_f3AmbientColor.y * g_fAmbientIntensity) * _f3BaseColor.y;
    //     _diffuseColor.z = (g_f3AmbientColor.z * g_fAmbientIntensity) * _f3BaseColor.z;
    // }

    return _diffuseColor;
}

template <int depth>
__device__ float3 material_compute_specular(SpecularMaterial& _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    float3 _sColor = make_float3(0, 0, 0);

    unsigned int _currentLightStreamSize = 0;
    uint32_t _lightStreamSize = _pLightStream->m_uiSize;

    while(_currentLightStreamSize < _lightStreamSize)
    {
        unsigned char* _curr = reinterpret_cast<unsigned char*>(_pLightStream);

        // LightStream* _lightStream = reinterpret_cast<LightStream*>(_curr);
        _curr += sizeof(LightStream) + _currentLightStreamSize;

        LightChunk* _chunk = reinterpret_cast<LightChunk*>(_curr);
        _curr += sizeof(LightChunk);

        if(_chunk->m_type == LightType::point_light)
        {
            PointLight* _light = reinterpret_cast<PointLight*>(_curr);
            _curr += sizeof(PointLight);

            float _intensity = 0.f;
            float3 _lightDir = normalize(_light->m_f3Location - _f3IntPoint);
            float3 _startPoint = _f3IntPoint + (_lightDir * 0.001f);
            Ray _lightRay(_startPoint, _startPoint + _lightDir);

            unsigned int _currentObjectStreamOffset = 0;
            uint32_t _objectStreamSize = _pObjectStream->m_uiSize;

            float3 _poi;
            float3 _poiNormal;
            float3 _poiColor;

            bool _intFound = false;

            ObjectChunk* _currentObject;

            while(_currentObjectStreamOffset < _objectStreamSize)
            {
                _intFound = test(_pObjectStream, _currentObjectStreamOffset, &_currentObject, _lightRay, _poi, _poiNormal, _poiColor);

                if(_intFound)
                {
                    break;
                }

                _currentObjectStreamOffset += sizeof(ObjectChunk) + _currentObject->m_uiSize;
            }

            if(!_intFound)
            {
                float3 _d = _lightRay.m_f3Lab;
                float3 _r = normalize(_d - (2.f * dot(_d, _f3LocalNormal) * _f3LocalNormal));
                float3 _v = normalize(_ray.m_f3Lab);
                float _dotProduct = dot(_r, _v);

                if(_dotProduct > 0.f)
                {
                    _intensity = _m.m_fReflectivity * pow(_dotProduct, _m.m_fShininess);
                }
            }

            _sColor += _light->m_f3Color * _intensity;
        }

        _currentLightStreamSize += sizeof(LightChunk) + sizeof(PointLight);
    }

    return _sColor;
}

template <>
__device__ float3 material_compute_specular<MAX_DEPTH>(SpecularMaterial& _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    return make_float3(0, 0, 0);
}

template <int depth>
__device__ float3 material_compute_reflection(SpecularMaterial* _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray);

__device__ bool material_cast_ray(const Ray& _ray, ObjectStream* _pObjectStream, ObjectChunk* _pCurrentObject, ObjectChunk** _ppClosestObject, Hit& _closestHit);

template <int depth>
__device__ float3 material_compute_color(const MaterialType& _materialType, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray);

template <int depth>
__device__ float3 material_compute_translucency(SpecularMaterial& _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    float3 _p = normalize(_ray.m_f3Lab);
    float3 _tmpNormal = _f3LocalNormal;
    float _r = 1.f / _m.m_fIOR;
    float _c = -dot(_tmpNormal, _p);

    if(_c < 0.f)
    {
        _tmpNormal = _tmpNormal * -1.f;
        _c = -dot(_tmpNormal, _p);
    }

    float3 _rV = _r * _p + (_r * _c - sqrtf(1.f - pow(_r, 2.f) * (1.f - pow(_c, 2.f)))) * _tmpNormal;

    Ray _rR(_f3IntPoint + (_rV * 0.01f), _f3IntPoint + _rV);

    float3 _poi;
    float3 _poiNormal;
    float3 _poiColor;
    ObjectChunk* _currentObject;
    ObjectChunk* _closestObject;
    Hit _closestHit;

    unsigned int _currentObjectStreamOffset = 0;
    uint32_t _objectStreamSize = _pObjectStream->m_uiSize;

    bool _validInt = false;
    bool _intFound = false;

    while(_currentObjectStreamOffset < _objectStreamSize)
    {
        if(test(_pObjectStream, _currentObjectStreamOffset, &_currentObject, _rR, _poi, _poiNormal, _poiColor))
        {
            _validInt = true;

            break;
        }

        _currentObjectStreamOffset += sizeof(ObjectChunk) + _currentObject->m_uiSize;
    }

    Ray _finalRay;

    if(_validInt)
    {
        float3 _p2 = normalize(_rR.m_f3Lab);
        float3 _tmpNormal2 = _poiNormal;
        float _r2 = 1.f / _m.m_fIOR;
        float _c2 = -dot(_tmpNormal, _p2);

        if(_c2 < 0.f)
        {
            _tmpNormal2 = _tmpNormal2 * -1.f;
            _c2 = -dot(_tmpNormal2, _p2);
        }

        float3 _rV2 = _r2 * _p2 + (_r2 * _c2 - sqrtf(1.f - pow(_r2, 2.f) * (1.f - pow(_c2, 2.f)))) * _tmpNormal2;

        Ray _rR2(_poi + (_rV2 * 0.01f), _poi + _rV2);

        _intFound = material_cast_ray(_rR2, _pObjectStream, _currentObject, &_closestObject, _closestHit);
        _finalRay = _rR2;
    }
    else
    {
        _intFound = material_cast_ray(_rR, _pObjectStream, _currentObject, &_closestObject, _closestHit);
        _finalRay = _rR;
    }

    if(_intFound)
    {
        if(_closestObject->m_material == MaterialType::none)
        {
            return material_compute_diffuse_color(_pObjectStream, _pLightStream, _closestObject, _closestHit.m_f3POI, _closestHit.m_f3POI_N, _closestHit.m_f3POI_C);
        }
        else
        {
            return material_compute_color<depth>(_closestObject->m_material, _pObjectStream, _pLightStream, _closestObject, _closestHit.m_f3POI, _closestHit.m_f3POI_N, _finalRay);
        }
    }

    return make_float3(0, 0, 0);
}

template <>
__device__ float3 material_compute_translucency<MAX_DEPTH>(SpecularMaterial& _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    return make_float3(0, 0, 0);
}

template <int depth>
__device__ float3 material_compute_specular(SpecularMaterial& _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    float3 _mColor = make_float3(0, 0, 0);
    float3 _rColor = make_float3(0, 0, 0);
    float3 _dColor = make_float3(0, 0, 0);
    float3 _sColor = make_float3(0, 0, 0);
    float3 _tColor = make_float3(0, 0, 0);

    _dColor = material_compute_diffuse_color(_pObjectStream, _pLightStream, _pCurrentObject, _f3IntPoint, _f3LocalNormal, _m.m_f3Color);

    if(_m.m_fReflectivity > 0.f)
    {
        _rColor = material_compute_reflection<depth+1>(&_m, _pObjectStream, _pLightStream, _pCurrentObject, _f3IntPoint, _f3LocalNormal, _ray);
    }

    _mColor = (_rColor * _m.m_fReflectivity) + (_dColor * (1.f - _m.m_fReflectivity));

    if(_m.m_fTranslucency > 0.f)
    {
        _tColor = material_compute_translucency<depth+1>(_m, _pObjectStream, _pLightStream, _pCurrentObject, _f3IntPoint, _f3LocalNormal, _ray);
    }

    _mColor = (_tColor * _m.m_fTranslucency) + (_mColor * (1.f - _m.m_fTranslucency));

    if(_m.m_fShininess > 0.f)
    {
        _sColor = material_compute_specular<depth+1>(_m, _pObjectStream, _pLightStream, _f3IntPoint, _f3LocalNormal, _ray);
    }

    return _mColor + _sColor;
}

template <>
__device__ float3 material_compute_specular<MAX_DEPTH>(SpecularMaterial& _m, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    return make_float3(0, 0, 0);
}

__device__ bool material_cast_ray(const Ray& _ray, ObjectStream* _pObjectStream, ObjectChunk* _pCurrentObject, ObjectChunk** _ppClosestObject, Hit& _closestHit)
{
    unsigned int _currentObjectStreamOffset = 0;
    uint32_t _objectStreamSize = _pObjectStream->m_uiSize;

    float _minDist = 1e6;
    bool _intFound = false;

    Hit _hit;

    ObjectChunk* _currentObject;

    while(_currentObjectStreamOffset < _objectStreamSize)
    {
        if(test(_pObjectStream, _currentObjectStreamOffset, &_currentObject, _ray, _hit.m_f3POI, _hit.m_f3POI_N, _hit.m_f3POI_C, _pCurrentObject))
        {
            _intFound = true;

            float _dist = norm(_hit.m_f3POI - _ray.m_f3Point1);

            if(_dist >= _minDist)
            {
                continue;
            }

            _minDist = _dist;
            *_ppClosestObject = _currentObject;
            _closestHit = _hit;
        }

        _currentObjectStreamOffset += sizeof(ObjectChunk) + _currentObject->m_uiSize;
    }

    return _intFound;
}

template <int depth>
__device__ float3 material_compute_reflection(SpecularMaterial* _pSpecularMaterial, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    float3 _d = _ray.m_f3Lab;
    float3 _r = _d - (2.f * dot(_d, _f3LocalNormal) * _f3LocalNormal);

    Ray _reflectionRay(_f3IntPoint, _f3IntPoint + _r);

    ObjectChunk* _closestObject;

    Hit _closestHit;

    if(!material_cast_ray(_reflectionRay, _pObjectStream, _pCurrentObject, &_closestObject, _closestHit))
    {
        return make_float3(0, 0, 0);
    }

    if(_closestObject->m_material == MaterialType::none)
    {
        return material_compute_diffuse_color(_pObjectStream, _pLightStream, _closestObject, _closestHit.m_f3POI, _closestHit.m_f3POI_N, _closestHit.m_f3POI_C);
    }
    else
    {
        return material_compute_color<depth>(_closestObject->m_material, _pObjectStream, _pLightStream, _closestObject, _closestHit.m_f3POI, _closestHit.m_f3POI_N, _reflectionRay);
    }

    return make_float3(0, 0, 0);
}

template <>
__device__ float3 material_compute_reflection<MAX_DEPTH>(SpecularMaterial* _pSpecularMaterial, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    return make_float3(0, 0, 0);
}

template <int depth>
__device__ float3 material_compute_color(const MaterialType& _materialType, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    if(_materialType == MaterialType::specular)
    {
        SpecularMaterial& _m = *reinterpret_cast<SpecularMaterial*>(reinterpret_cast<uint8_t*>(_pCurrentObject) + sizeof(ObjectChunk));

        return material_compute_specular<depth>(_m, _pObjectStream, _pLightStream, _pCurrentObject, _f3IntPoint, _f3LocalNormal, _ray);
    }

    return make_float3(0, 0, 0);
}

template <>
__device__ float3 material_compute_color<MAX_DEPTH>(const MaterialType& _materialType, ObjectStream* _pObjectStream, LightStream* _pLightStream, ObjectChunk* _pCurrentObject, const float3& _f3IntPoint, const float3& _f3LocalNormal, const Ray& _ray)
{
    return make_float3(0, 0, 0);
}

__global__ void d_ray_trace(unsigned int* _aPixels, unsigned int _uiWidth, unsigned int _uiHeight, Camera& _camera, ObjectStream* _pObjectStream, LightStream* _pLightStream)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i >= _uiWidth || j >= _uiHeight)
    {
		return;
    }

    float x = (float)i;
	float y = (float)j;

    Ray _ray;

    float3 _intPoint;
    float3 _localNormal;
    float3 _localColor;

    float xFact = 1.f / ((float)_uiWidth / 2.f);
    float yFact = 1.f / ((float)_uiHeight / 2.f);

    float normX = ((float)x * xFact) - 1.f;
    float normY = ((float)y * yFact) - 1.f;

    generate_ray(_camera, _ray, normX, normY);

    float3 _closestIntPoint = make_float3(0, 0, 0);
    float3 _closestLocalNormal = make_float3(0, 0, 0);
    float3 _closestLocalColor = make_float3(0, 0, 0);

    ObjectChunk* _closestObject;

    float3 pixelColor = make_float3(0, 0, 0);

    if(find_closest(_pObjectStream, _ray, &_closestObject, _closestIntPoint, _closestLocalNormal, _closestLocalColor))
    {
        if(_closestObject->m_material == MaterialType::none)
        {
            pixelColor = material_compute_diffuse_color(_pObjectStream, _pLightStream, _closestObject, _closestIntPoint, _closestLocalNormal, _closestLocalColor);
        }
        else
        {
            pixelColor = material_compute_color<0>(_closestObject->m_material, _pObjectStream, _pLightStream, _closestObject, _closestIntPoint, _closestLocalNormal, _ray);
        }
    }

    _aPixels[j * _uiWidth + i] = rgbToInt(pixelColor.x, pixelColor.y, pixelColor.z);
}

extern "C" __declspec(dllexport) void ray_trace(unsigned int* _aPixels, unsigned int _uiWidth, unsigned int _uiHeight, Camera* _pCamera, ObjectStream* _pObjectStream, LightStream* _pLightStream)
{
    // int blockSize = 256;
    // int numBlocks = (n + blockSize - 1) / blockSize;

    dim3 block(16, 16, 1);
	dim3 grid(ceil((float)_uiWidth / block.x), ceil((float)_uiHeight / block.y), 1);

    d_ray_trace<<<grid, block>>>(_aPixels, _uiWidth, _uiHeight, *_pCamera, _pObjectStream, _pLightStream);

    auto _e = hipGetLastError();

    if(_e == hipSuccess)
    {
        return;
    }

    printf("%s\n", hipGetErrorString(_e));
}